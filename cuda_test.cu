#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  printf("hipMemcpyAsync addr: %p\n", hipMemcpyAsync);
  hipStream_t stream1;
  hipStreamCreate(&stream1);
 
  int N = 1<<20;
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpyAsync(d_x, x, N*sizeof(float), hipMemcpyHostToDevice, stream1);
  hipMemcpyAsync(d_y, y, N*sizeof(float), hipMemcpyHostToDevice, stream1);

  // Perform SAXPY on 1M elements
  //saxpy<<<(N+255)/256, 256, 0, stream1>>>(N, 2.0f, d_x, d_y);
  unsigned int blocks = (N+255)/256;
  unsigned int threads = 256;
  float a = 2.0f;
  void *params[] = { (void*)&N, (void*)&a, (void*)&d_x, (void*)&d_y };
  hipLaunchKernel((void*)saxpy, dim3(blocks), dim3(threads), params, 0, stream1);

  hipMemcpyAsync(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost, stream1);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = max(maxError, abs(y[i]-4.0f));
  printf("Max error: %f\n", maxError);

  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
  hipStreamDestroy(stream1);
}
